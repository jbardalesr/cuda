
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

using namespace std;

// create a kernel to estimate pi
__global__ void count_samples_in_circles(float *d_randNumsX, float *d_randNumsY, int *d_countInBlocks, int num_blocks, int nsamples)
{

    __shared__ int shared_blocks[500];

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * num_blocks;

    // iterates through
    int inCircle = 0;
    for (int i = tid; i < nsamples; i += stride)
    {
        float xValue = d_randNumsX[i];
        float yValue = d_randNumsY[i];

        if (xValue * xValue + yValue * yValue <= 1.0f)
        {
            inCircle++;
        }
    }

    shared_blocks[threadIdx.x] = inCircle;

    __syncthreads();

    // pick thread 0 for each block to collect all points from each Thread.
    if (threadIdx.x == 0)
    {
        int totalInCircleForABlock = 0;
        for (int j = 0; j < blockDim.x; j++)
        {
            totalInCircleForABlock += shared_blocks[j];
        }
        d_countInBlocks[blockIdx.x] = totalInCircleForABlock;
    }
}

int nsamples = 1e8;

int main(void)
{
    // allocate space to hold random values
    vector<float> h_randNumsX(nsamples);
    vector<float> h_randNumsY(nsamples);
    srand(time(NULL)); // seed with system clock

    // initialize vector with random values
    for (int i = 0; i < h_randNumsX.size(); ++i)
    {
        h_randNumsX[i] = float(rand()) / RAND_MAX;
        h_randNumsY[i] = float(rand()) / RAND_MAX;
    }

    // send random values to the GPU
    size_t size = nsamples * sizeof(float);
    float *d_randNumsX;
    float *d_randNumsY;

    hipMalloc(&d_randNumsX, size);
    hipMalloc(&d_randNumsY, size);
    hipMemcpy(d_randNumsX, &h_randNumsX.front(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_randNumsY, &h_randNumsY.front(), size, hipMemcpyHostToDevice);

    // launch kernel to count samples that fell inside unit circle
    int threadsPerBlock = 500;
    int num_blocks = nsamples / (1000 * threadsPerBlock);
    size_t countBlocks = num_blocks * sizeof(int);

    // number of parallel blocks
    int *d_countInBlocks;
    hipMalloc(&d_countInBlocks, countBlocks);

    // call kernel
    count_samples_in_circles<<<num_blocks, threadsPerBlock>>>(d_randNumsX, d_randNumsY, d_countInBlocks, num_blocks, nsamples);
    if (hipSuccess != hipGetLastError())
        cout << "Error!\n";

    // return back the vector from device to host
    int *h_countInBlocks = new int[num_blocks];
    hipMemcpy(h_countInBlocks, d_countInBlocks, countBlocks, hipMemcpyDeviceToHost);

    int nsamples_in_circle = 0;
    for (int i = 0; i < num_blocks; i++)
    {
        // cout << "Value in block " + i << " is " << h_countInBlocks[i] << endl;
        nsamples_in_circle = nsamples_in_circle + h_countInBlocks[i];
    }

    hipFree(d_randNumsX);
    hipFree(d_randNumsY);
    hipFree(d_countInBlocks);

    // fraction that fell within (quarter) of unit circle
    float estimatedValue = 4.0 * float(nsamples_in_circle) / nsamples;

    cout << "Estimated Value: " << estimatedValue << endl;
}
