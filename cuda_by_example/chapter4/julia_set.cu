#include "../common/book.h"
#include "../common/cpu_bitmap.h"

#define DIM 10

struct hipComplex
{
    float r;
    float i;
    hipComplex(float a, float b) : r(a), i(b) {}

    float magnitude2(void)
    {
        return r * r + i * i;
    }

    hipComplex operator*(const hipComplex &a)
    {
        return hipComplex(r * a.r - i * a.i, i * a.r + r * a.i);
    }

    hipComplex operator+(const hipComplex &a)
    {
        return hipComplex(r + a.r, i + a.i);
    }
};

int julia(int x, int y)
{
    const float scale = 1.5;
    float jx = scale * (float)(DIM / 2 - x) / (DIM / 2);
    float jy = scale * (float)(DIM / 2 - y) / (DIM / 2);

    hipComplex c(-0.8, 0.156);
    hipComplex a(jx, jy);

    int i = 0;
    for (i = 0; i < 200; i++)
    {
        a = a * a + c;
        if (a.magnitude2() > 1000)
            return 0;
    }
}

void kernel(unsigned char *ptr)
{
    for (int y = 0; y < DIM; y++)
    {
        for (int x = 0; x < DIM; x++)
        {
            int offset = x + y * DIM;

            int juliaValue = julia(x, y);
            ptr[offset * 4 + 0] = 255 * juliaValue;
            ptr[offset * 4 + 1] = 0;
            ptr[offset * 4 + 2] = 0;
            ptr[offset * 4 + 3] = 255;
        }
    }
}

int main(void)
{
    CPUBitmap bitmap(DIM, DIM);
    unsigned char *dev_bitmap;

    HANDLE_ERROR(hipMalloc((void **)&dev_bitmap, bitmap.image_size()));
    dim3 grid(DIM, DIM);

    kernel<<<grid, 1>>>(dev_bitmap);

    HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(),
                            dev_bitmap,
                            bitmap.image_size(),
                            hipMemcpyDeviceToHost));

    bitmap.display_and_exit();
    hipFree(dev_bitmap);
}