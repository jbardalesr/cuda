#include "hip/hip_runtime.h"
/*
Example 3.2.3

- Define size of array
- Inside main function
    - Allocate the memory on the GPU with hipMalloc
    - Fill the arrays 'a' and 'b' on the CPU
    - Copy the arrays 'a' and 'b' on the GPU with cudaMemCpy() and type hipMemcpyHostToDevice
    - Apply add function
    - Copy the array 'c' back from the GPU to the CPU
    - Display the results
    - Free the memory allocated on the GPU
*/

#include "../common/book.h"

#define N 10

__global__ void add(int *a, int *b, int *c)
{
    // blockIdx contains the value of the block  index for whichever block is currently running the device code
    int tid = blockIdx.x; // handle the data at this index
    if (tid < N)
    {
        c[tid] = a[tid] + b[tid];
    }
}

int main(void)
{
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    // allocate the memory on the GPU
    HANDLE_ERROR(hipMalloc((void **)&dev_a, sizeof(int) * N));
    HANDLE_ERROR(hipMalloc((void **)&dev_b, sizeof(int) * N));
    HANDLE_ERROR(hipMalloc((void **)&dev_c, sizeof(int) * N));

    // fill the arrays 'a' and 'b' on the CPU
    for (int i = 0; i < N; i++)
    {
        a[i] = -i;
        b[i] = i * i;
    }

    // copy the arrays 'a' and 'b' on the GPU
    HANDLE_ERROR(hipMemcpy(dev_a, a, sizeof(int) * N, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, sizeof(int) * N, hipMemcpyHostToDevice));

    // execute the device code
    add<<<N, 1>>>(dev_a, dev_b, dev_c);

    // copy the array 'c' back from the GPU to the CPU
    HANDLE_ERROR(hipMemcpy(c, dev_c, sizeof(int) * N, hipMemcpyDeviceToHost));

    // display the results
    for (int i = 0; i < N; i++)
    {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    // free the memory allocated on the GPU
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}