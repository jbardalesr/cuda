#include "hip/hip_runtime.h"
/*
Example 3.2.3

- Declare a global function calls add
- Inside main function
    - Declare a pointer to device calls dev_c
    - Allocate memory in device with hipMalloc()
    - Pass the parameters to add function
    - Copy result into device with cudaMemCpy() and type hipMemcpyDeviceToHost
    - Print result
    - Free Memory hipFree()
*/

#include <stdio.h>
#include "../common/book.h"

__global__ void add(int a, int b, int *c)
{
    *c = a * b;
}

int main(void)
{
    int c;
    // device pointer
    int *dev_c;
    // hipMalloc: allocate memory on the device
    HANDLE_ERROR(hipMalloc((void **)&dev_c, sizeof(int)));
    // pass the parameters
    add<<<1, 1>>>(2, 7, dev_c);
    // hipMemcpy: copies data between host and device.
    HANDLE_ERROR(hipMemcpy(&c,
                            dev_c,
                            sizeof(int),
                            hipMemcpyDeviceToHost));
    printf("2 * 7 = %d\n", c);

    int count;
    HANDLE_ERROR(hipGetDeviceCount(&count));
    printf("Number of devices = %d\n", count);

    // hipFree: frees memory on the device.
    hipFree(&dev_c);
    return 0;
}
